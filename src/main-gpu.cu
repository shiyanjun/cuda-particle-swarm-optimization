#include "hip/hip_runtime.h"
/*
Particle swarm optimization
by Ivan Vinogradov
2016
*/

#include <iostream>
#include <chrono>
#include <cfloat>
#include <cmath>

// OpenGL
#include <GL/glew.h>
#include <GL/freeglut.h>

// CUDA
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// cuRAND
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

// Thrust
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>

#include "common.cuh"
#include "particle.cuh"

// Window size, center, zoom, function min-max, time
__constant__ int devWidth;
__constant__ int devHeight;
__constant__ int devParticleSize;
__constant__ double devCenterX;
__constant__ double devCenterY;
__constant__ double devZoomX;
__constant__ double devZoomY;
__constant__ double devTimeValue;
__constant__ double devTimeStep;

// Parameters of uniform space partitioning
__constant__ double devUniformSpaceMinX;
__constant__ double devUniformSpaceMaxX;
__constant__ double devUniformSpaceMinY;
__constant__ double devUniformSpaceMaxY;
__constant__ double devUniformSpaceCellSize;

Particle *devParticleArray;
ParticleArea *devPartileAreaArray;
hiprandState *devRandomState;

// Screen coordinates into real coordinates
__device__
double2 indexToCoord(int2 index) {
  return make_double2(
    (2.0f * index.x / (double)(devWidth - 1) - 1.0f) * devZoomX + devCenterX,
    -(2.0f * index.y / (double)(devHeight - 1) - 1.0f) * devZoomY + devCenterY);
}

// Real coordinates into screen coordinates
__device__
int2 coordToIndex(double2 coord) {
  return make_int2(
    0.5f * (devWidth - 1) * (1.0f + (coord.x - devCenterX) / devZoomX),
    0.5f * (devHeight - 1) * (1.0f - (coord.y - devCenterY) / devZoomY)
  );
}

/*  Schwefel Function */
__device__
double fun(double2 coord) {
  return -coord.x * sin(sqrt(fabs(coord.x))) - coord.y * sin(sqrt(fabs(coord.y)));
}

__device__
double fun(int2 index) {
  return fun(indexToCoord(index));
}

__global__
void initRandomState(hiprandState *state, int n) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int offsetx = blockDim.x * gridDim.x;
  for (int i = idx; i < n; i += offsetx) {
    hiprand_init(1337, i, 0, &state[i]);
  }
}

__global__
void kernelSwarmInit(Particle *particles, int n, hiprandState *state) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int offsetx = blockDim.x * gridDim.x;
  Particle *p;
  for (int i = idx; i < n; i += offsetx) {
    p = &particles[i];

    // Position in the center of the screen
    p->coords = p->best_coords = indexToCoord(make_int2(devWidth / 2, devHeight / 2));

    // // Random position within the screen
    // p->coords = p->best_coords = indexToCoord(make_int2(
    //   hiprand_uniform_double(&state[i]) * devWidth,
    //   hiprand_uniform_double(&state[i]) * devHeight
    // ));

    // Random starting angle and the speed
    double angle = 2.0 * 3.14 * hiprand_uniform_double(&state[i]);
    double speed = 100.0 * hiprand_uniform_double(&state[i]);
    p->speed = make_double2(cos(angle) * speed, sin(angle) * speed);
    p->value = p->best_value = DBL_MAX;
  }
}

__global__
void kernelSwarmUpdate(uchar4 *image, Particle *particles, int n) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int offsetx = blockDim.x * gridDim.x;

  Particle *p;

  for (int i = idx; i < n; i += offsetx) {
    p = &particles[i];
    p->value = fun(p->coords);
    if (p->value < p->best_value) {
      p->best_value = p->value;
      p->best_coords = p->coords;
    }
  }
}

__global__
void kernelNormalizedHeatMap(uchar4 *heatMap, double minValue, double maxValue) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y + threadIdx.y;
  int offsetx = blockDim.x * gridDim.x;
  int offsety = blockDim.y * gridDim.y;
  int i, j;
  double f;
  for (i = idx; i < devWidth; i += offsetx) {
    for (j = idy; j < devHeight; j += offsety) {
      f = (fun(make_int2(i, j)) - minValue) / (maxValue - minValue);
      if (f < 0.0) f = 0.0; else if (f > 1.0) f = 1.0;
      heatMap[j * devWidth + i] = make_uchar4(
        (int)(f * 255), 0, (int)((1.0 - f) * 255), 255
      );
    }
  }
}

__global__
void kernelSwarmDraw(uchar4 *heatMap, Particle *particles, int n) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int offsetx = blockDim.x * gridDim.x;
  int size = devParticleSize;
  for (int i = idx; i < n; i += offsetx) {
    int2 index = coordToIndex(particles[i].coords);
    for (int x = index.x - size; x <= index.x + size; x++) {
      for (int y = index.y - size; y <= index.y + size; y++) {
        if (x >= 0 && x < devWidth && y >= 0 && y < devHeight &&
            (x - index.x) * (x - index.x) + (y - index.y) * (y - index.y) <= size * size
        ) {

          if (DEBUG && i == n / 2) {
            heatMap[y * devWidth + x] = make_uchar4(0, 255, 0, 255);
            continue;
          }

          heatMap[y * devWidth + x] = make_uchar4(255, 255, 255, 255);
        }
      }
    }
  }
}

// The assignment for each particle corresponding space partitioning cell
__global__
void kernelSwarmAssociateWithCells(Particle *particles, int n) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int offsetx = blockDim.x * gridDim.x;

  int sizeX = ceil((devUniformSpaceMaxX - devUniformSpaceMinX) / devUniformSpaceCellSize);
  int sizeY = ceil((devUniformSpaceMaxY - devUniformSpaceMinY) / devUniformSpaceCellSize);

  Particle *p;
  int cellX, cellY;

  for (int i = idx; i < n; i += offsetx) {
    p = &particles[i];
    cellX = (p->coords.x - devUniformSpaceMinX) / devUniformSpaceCellSize;
    cellY = (p->coords.y - devUniformSpaceMinY) / devUniformSpaceCellSize;
    p->cellIndex = cellX * sizeX + cellY;
  }
}

// The total force of repulsion for the i-th particle (without the space partitioning)
__device__
double2 calculateRepulsionAll(Particle *particles, int n, int i) {
  double2 repulsion, diff, coords_a, coords_b;
  double distance;

  // TODO
  double minDistance = DBL_MAX;

  repulsion.x = 0.0;
  repulsion.y = 0.0;
  coords_a = (&particles[i])->coords;

  for (int j = 0; j < n; j++) {
    if (j == i) continue;
    coords_b = (&particles[j])->coords;
    diff.x = coords_a.x - coords_b.x;
    diff.y = coords_a.y - coords_b.y;
    distance = sqrt(diff.x * diff.x + diff.y * diff.y);

    // TODO
    if (DEBUG && i == n / 2 && distance < minDistance) {
      minDistance = distance;
    }

    distance = pow(distance, 5);

    if (distance < 0.5) distance = 0.5;

    repulsion.x += diff.x / distance;
    repulsion.y += diff.y / distance;
  }

  // TODO
  if (DEBUG && i == n / 2) {
    printf("distance: %lf; interactions: %d\n", minDistance, n);
  }
  
  // TODO
  // repulsion.x /= n;
  // repulsion.y /= n;
  return repulsion;
}

// Binary search in a sorted array of particles by cell index of space partitioning
__device__
int binarySearchLowerBound(Particle *particles, int size, int cellIndex) {
    int left = 0, right = size - 1, middle;
    while (left <= right) {
        middle = left + (right - left) / 2;
        if ((&particles[middle])->cellIndex < cellIndex)
            left = middle + 1;
        else
            right = middle - 1;
    }
    return left;
}

// Check that the particle was found in binarySearchLowerBound
__device__
int isFound(Particle *particles, int size, int cellIndex, int index) {
    return index < size && particles[index].cellIndex == cellIndex;
}

// Search the first particle with specified cell partition index
__device__
int findParticleByCell(Particle *particles, int size, int cellIndex) {
  int index = binarySearchLowerBound(particles, size, cellIndex);
  return isFound(particles, size, cellIndex, index) ? index : -1;
}

// The total force of repulsion for the i-th particle (with the space partitioning)
__device__
double2 calculateRepulsionClosest(Particle *particles, int n, int i) {
  double2 diff, repulsion = make_double2(0.0, 0.0);
  double distance;

  // TODO
  double minDistance = DBL_MAX;

  // Counter of interacting particles
  int counter = 0;

  // Dimensions of the space partitioning
  int sizeX = ceil(abs(devUniformSpaceMaxX - devUniformSpaceMinX) / devUniformSpaceCellSize);
  int sizeY = ceil(abs(devUniformSpaceMaxY - devUniformSpaceMinY) / devUniformSpaceCellSize);

  if (sizeX < 1) sizeX = 1;
  if (sizeY < 1) sizeY = 1;

  Particle *pa, *pb;

  pa = &particles[i];

  int cellIndex = pa->cellIndex;

  // TODO: деление на ноль
  int cellX = cellIndex / sizeX;
  int cellY = cellIndex % sizeX;

  int radius = 1;
  for (int x = cellX - radius; x <= cellX + radius; x++) {
    for (int y = cellY - radius; y <= cellY + radius; y++) {
      int neighborCellIndex = x * sizeX + y;
      int neighborIndex = findParticleByCell(particles, n, neighborCellIndex);
      if (neighborIndex != -1)
      {
        for (int k = neighborIndex; k < n; k++)
        {
          if (k == i) continue;
          pb = &particles[k];
          if (pb->cellIndex != neighborCellIndex) break;

          diff.x = pa->coords.x - pb->coords.x;
          diff.y = pa->coords.y - pb->coords.y;

          distance = sqrt(diff.x * diff.x + diff.y * diff.y);

          // TODO
          if (DEBUG && i == n / 2 && distance < minDistance) {
            minDistance = distance;
          }

          distance = pow(distance, 5);

          if (distance < 0.5) distance = 0.5;

          repulsion.x += diff.x / distance;
          repulsion.y += diff.y / distance;

          counter++;
        }
      }
    }
  }

  // TODO
  if (DEBUG && i == n / 2 && counter > 0) {
    printf("distance: %lf; interactions: %d\n", minDistance, counter);
  }

  // TODO
  // if (counter > 1) {
  //   repulsion.x /= counter;
  //   repulsion.y /= counter;
  // }

  return repulsion;
}

__global__
void kernelSwarmMove(uchar4 *image, Particle *particles, int n,
                     double2 global_minimum, hiprandState *state) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int offsetx = blockDim.x * gridDim.x;

  const double g_coeff = 0.000050;     // coefficient of global solution
  const double p_coeff = 0.00000010;   // coefficient of local solution
  const double rnd_coeff = 0.010;      // coefficient of random motion
  const double damping_coeff = 0.991;  // coefficient of damping force
  const double repulsion_coeff = 0.10; // coefficient of repulsive force

  // The speed drops to 0.1 depending on the time
  double speed_coeff = 1.0 / (1.0 + 0.01 * pow(devTimeValue, 4));
  speed_coeff = speed_coeff < 0.1 ? 0.1 : speed_coeff;

  double rnd_1, rnd_2, rnd_3, rnd_4;
  Particle *p;
  double2 repulsion;

  for (int i = idx; i < n; i += offsetx) {
    rnd_1 = hiprand_uniform_double(&state[i]);
    rnd_2 = hiprand_uniform_double(&state[i]);
    rnd_3 = hiprand_uniform_double(&state[i]);
    rnd_4 = hiprand_uniform_double(&state[i]);

    p = &particles[i];

    p->speed.x = damping_coeff * p->speed.x +
                 speed_coeff * (
                   rnd_1 * p_coeff * (p->best_coords.x - p->coords.x) +
                   rnd_2 * g_coeff * (global_minimum.x - p->coords.x) +
                   rnd_coeff * (rnd_3 - 0.5)
                 );

    p->speed.y = damping_coeff * p->speed.y +
                 speed_coeff * (
                   rnd_1 * p_coeff * (p->best_coords.y - p->coords.y) +
                   rnd_2 * g_coeff * (global_minimum.y - p->coords.y) +
                   rnd_coeff * (rnd_4 - 0.5)
                 );

    // repulsion = calculateRepulsionAll(particles, n, i);
    repulsion = calculateRepulsionClosest(particles, n, i);
    p->speed.x += repulsion_coeff * repulsion.x;
    p->speed.y += repulsion_coeff * repulsion.y;

    p->coords.x += p->speed.x;
    p->coords.y += p->speed.y;
  }
}

void copySizeToGPU() {
  CSC(hipMemcpyToSymbol((const void *)&devWidth, &width, sizeof(int)));
  CSC(hipMemcpyToSymbol((const void *)&devHeight, &height, sizeof(int)));
}

void copyZoomToGPU() {
  CSC(hipMemcpyToSymbol((const void *)&devZoomX, &zoomX, sizeof(double)));
  CSC(hipMemcpyToSymbol((const void *)&devZoomY, &zoomY, sizeof(double)));
}

void copyParticleSizeToGPU() {
  CSC(hipMemcpyToSymbol((const void *)&devParticleSize, &particleSize, sizeof(int)));
}

void copyCenterToGPU() {
  CSC(hipMemcpyToSymbol((const void *)&devCenterX, &centerX, sizeof(double)));
  CSC(hipMemcpyToSymbol((const void *)&devCenterY, &centerY, sizeof(double)));
}

void copyTimeToGPU() {
  CSC(hipMemcpyToSymbol((const void *)&devTimeValue, &timeValue, sizeof(double)));
  CSC(hipMemcpyToSymbol((const void *)&devTimeStep, &timeStep, sizeof(double)));
}

void copyUniformSpaceToGPU(double minX, double maxX, double minY, double maxY, double cellSize) {
  CSC(hipMemcpyToSymbol((const void *)&devUniformSpaceMinX, &minX, sizeof(double)));
  CSC(hipMemcpyToSymbol((const void *)&devUniformSpaceMaxX, &maxX, sizeof(double)));
  CSC(hipMemcpyToSymbol((const void *)&devUniformSpaceMinY, &minY, sizeof(double)));
  CSC(hipMemcpyToSymbol((const void *)&devUniformSpaceMaxY, &maxY, sizeof(double)));
  CSC(hipMemcpyToSymbol((const void *)&devUniformSpaceCellSize, &cellSize, sizeof(double)));
}

void copyToGPU() {
  copySizeToGPU();
  copyZoomToGPU();
  copyParticleSizeToGPU();
  copyCenterToGPU();
  copyTimeToGPU();
}

void update() {

  auto t_start = std::chrono::high_resolution_clock::now();

  copyToGPU();

  uchar4 *devHeatMap;
  size_t size;

  CSC(hipGraphicsMapResources(1, &res, 0));
  CSC(hipGraphicsResourceGetMappedPointer((void **)&devHeatMap, &size, res));

  // Update the function values and local minima for each particle
  kernelSwarmUpdate<<<blocks_1d, threads_1d>>>(devHeatMap, devParticleArray, numberOfParticles);

  // The boundaries and the center of the swarm, minimum, maximum, the global minimum
  thrust::device_ptr<ParticleArea> startParticleAreaArray(devPartileAreaArray);
  thrust::device_ptr<ParticleArea> endParticleAreaArray = startParticleAreaArray + numberOfParticles;

  ParticleArea pa;
  pa.min_x = DBL_MAX ;
  pa.min_y = DBL_MAX ;
  pa.max_x = -DBL_MAX ;
  pa.max_y = -DBL_MAX ;
  pa.sum_x = 0.0;
  pa.sum_y = 0.0;
  pa.minValue = DBL_MAX ;
  pa.maxValue = -DBL_MAX ;
  pa.globalMinimum = DBL_MAX ;

  kernelInitParticleArea<<<blocks_1d, threads_1d>>>(
    devParticleArray, devPartileAreaArray, numberOfParticles
  );
  pa = thrust::reduce(startParticleAreaArray, endParticleAreaArray, pa, ParticleReductionFunctor());

  // Align the window in the center of particle swarm
  if (autoCenter) {
      centerX = pa.sum_x / numberOfParticles;
      centerY = pa.sum_y / numberOfParticles;
      copyCenterToGPU();
  }

  // Draw a heat map and particles
  kernelNormalizedHeatMap<<<blocks_2d, threads_2d>>>(devHeatMap, pa.minValue, pa.maxValue);
  kernelSwarmDraw<<<blocks_1d, threads_1d>>>(devHeatMap, devParticleArray, numberOfParticles);

  // Space partitioning
  copyUniformSpaceToGPU(pa.min_x, pa.max_x, pa.min_y, pa.max_y, cellSize);
  kernelSwarmAssociateWithCells<<<blocks_1d, threads_1d>>>(devParticleArray, numberOfParticles);

  // Sort particles by cell index
  thrust::device_ptr<Particle> startParticleArray(devParticleArray);
  thrust::device_ptr<Particle> endParticleArray = startParticleArray + numberOfParticles;
  thrust::sort(startParticleArray, endParticleArray, ParticleSortByIndexComparator());

  // Update particles position
  kernelSwarmMove<<<blocks_1d, threads_1d>>>(
    devHeatMap, devParticleArray, numberOfParticles, pa.globalMinimumCoords, devRandomState
  );

  timeValue += timeStep;

  CSC(hipDeviceSynchronize());
  CSC(hipGraphicsUnmapResources(1, &res, 0));

  auto t_end = std::chrono::high_resolution_clock::now();
  double duration = std::chrono::duration<double, std::milli>(t_end-t_start).count();
  if (DEBUG) {
    printf("%lf ms; center: %lf, %lf\n\n", duration, centerX, centerY);
  }

  glutPostRedisplay();
}

void display() {
  glMatrixMode(GL_PROJECTION);
  glLoadIdentity();
  gluOrtho2D(0.0, (GLdouble)width, 0.0, (GLdouble)height);
  glutInitWindowSize(width, height);
  glClearColor(0.0, 0.0, 0.0, 1.0);
  glClear(GL_COLOR_BUFFER_BIT);
  glDrawPixels(width, height, GL_RGBA, GL_UNSIGNED_BYTE, 0);
  glutSwapBuffers();
}

int main(int argc, char **argv) {

  std::cout << "Enter window width: ";
  std::cin >> width;

  std::cout << "Enter window height: ";\
  std::cin >> height;

  std::cout << "Enter number of particles: ";
  std::cin >> numberOfParticles;

  std::cout << "Enter cell size: ";
  std::cin >> cellSize;

  copyToGPU();

  hipMalloc((void **)&devRandomState, sizeof(hiprandState) * numberOfParticles);
  hipMalloc((void **)&devParticleArray, sizeof(Particle) * numberOfParticles);
  hipMalloc((void **)&devPartileAreaArray, sizeof(Particle) * numberOfParticles);

  initRandomState<<<blocks_1d, threads_1d>>>(devRandomState, numberOfParticles);
  kernelSwarmInit<<<blocks_1d, threads_1d>>>(devParticleArray, numberOfParticles, devRandomState);

  glutInit(&argc, argv);
  glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);
  glutInitWindowSize(width, height);
  glutCreateWindow("Particle swarm optimization");

  glutIdleFunc(update);
  glutDisplayFunc(display);
  glutReshapeFunc(reshapeFunc);
  glutKeyboardFunc(keyboardFunc);
  glutMouseWheelFunc(mouseWheelFunc);

  glMatrixMode(GL_PROJECTION);
  glLoadIdentity();
  gluOrtho2D(0.0, (GLdouble)width, 0.0, (GLdouble)height);

  glewInit();

  GLuint vbo;
  glGenBuffers(1, &vbo);
  glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, vbo);
  glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, width * height * sizeof(uchar4), NULL, GL_DYNAMIC_DRAW);

  CSC(hipGraphicsGLRegisterBuffer(&res, vbo, cudaGraphicsMapFlagsWriteDiscard));

  glutMainLoop();

  CSC(hipGraphicsUnregisterResource(res));

  glBindBuffer(1, vbo);
  glDeleteBuffers(1, &vbo);

  hipFree(devParticleArray);
  hipFree(devRandomState);

  return 0;
}
